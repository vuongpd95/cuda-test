/******************************************************************************
* PROGRAM: copyStruture
* PURPOSE: This program is a test which test the ability to transfer multilevel 
*	C++ structured data from host to device, modify them and transfer back.
*
*
* NAME: Vuong Pham-Duy.
*	College student.
*       Faculty of Computer Science and Technology.
*       Ho Chi Minh University of Technology, Viet Nam.
*       vuongpd95@gmail.com
*
* DATE: 5/10/2017
*
******************************************************************************/

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", \
			hipGetErrorString(code), file, line);
		if (abort) 
			exit(code);
	}
}

__global__ void func(int *value) {
	printf("value[%d] = %d\nvalue[%d] = %d\n", \
		value[0], value[0], value[1], value[1]);
}

int main(int argc, char *argv[])
{
	int *value;
	gpuErrchk(hipMallocManaged(&value, 2 * sizeof(int)));
	value[0] = 0;
	value[1] = 1;
	func<<<1, 1>>>(value);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipFree(value));
	return 0;
}
