#include "hip/hip_runtime.h"
/******************************************************************************
* PROGRAM: copyStruture
* PURPOSE: This program is a test which test the ability to transfer multilevel 
*	C++ structured data from host to device, modify them and transfer back.
*
*
* NAME: Vuong Pham-Duy.
*	College student.
*       Faculty of Computer Science and Technology.
*       Ho Chi Minh University of Technology, Viet Nam.
*       vuongpd95@gmail.com
*
* DATE: 5/10/2017
*
******************************************************************************/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#define WARP 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", \
			hipGetErrorString(code), file, line);
		if (abort) 
			exit(code);
	}
}
/* Data structures */
typedef struct {
	int32_t h, e;
} eh_t;

extern __shared__ eh_t s_qp[];

__global__ 
void func_0(int qlen, eh_t *d_qp) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int lane_id = threadIdx.x % 32;
	eh_t data;
	int beg = 0, f = 0;
	if(lane_id == 0) {
		memcpy(s_qp, d_qp, sizeof(int8_t) * (qlen + 1));
	}
	do {
	} while(beg < qlen);
}

int main(int argc, char *argv[])
{
	int qlen;
	printf("Input qlen = ");
	scanf("%d", &qlen);

	int8_t *h_qp, *d_qp;
	h_qp = (int8_t*)malloc(sizeof(eh_t) * (qlen + 1)));
	int k = 0;
	for(int i = 0; i <= qlen; i++) {
		h_qp[i].h = h_qp[i].e = i;
	}

	gpuErrchk(hipMalloc(&d_qp, sizeof(eh_t) * (qlen + 1)));
	gpuErrchk(hipMemcpy(d_qp, h_qp, sizeof(eh_t) * (qlen + 1), hipMemcpyHostToDevice));


	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventRecord(start, 0);

	func_0<<<1, WARP, qlen + 1>>>(qlen, d_qp);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	fprintf(stderr, "[M::%s] Kernel executed in %f ms\n" , __func__, elapsedTime);

	return 0;
	
}
