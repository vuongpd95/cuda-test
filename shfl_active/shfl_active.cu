#include "hip/hip_runtime.h"
/******************************************************************************
* PROGRAM: copyStruture
* PURPOSE: This program is a test which test the ability to transfer multilevel 
*	C++ structured data from host to device, modify them and transfer back.
*
*
* NAME: Vuong Pham-Duy.
*	College student.
*       Faculty of Computer Science and Technology.
*       Ho Chi Minh University of Technology, Viet Nam.
*       vuongpd95@gmail.com
*
* DATE: 5/10/2017
*
******************************************************************************/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#define WARP 32
#define LANE 2
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", \
			hipGetErrorString(code), file, line);
		if (abort) 
			exit(code);
	}
}
/* Data structures */
typedef struct {
	int32_t h, e;
} eh_t;

__device__
bool check_active(eh_t data) {
	if(data.h != -1 && data.e != -1) return true;
	else return false;
}
__device__
void reset(eh_t *data) {
	data->h = -1;
	data->e = -1;
}
__global__ 
void func_0(int qlen, eh_t *d_qp) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int lane_id = threadIdx.x % 32;
	eh_t data_out, data_in;
	reset(&data_in); reset(&data_out);
	int beg = 0;
	if(lane_id == LANE) printf("Lane %d, received: ");
	if(lane_id == 0) memcpy(s_qp, d_qp, sizeof(int8_t) * (qlen + 1));
	if(lane_id == 0) {
		data_out = s_qp[beg];
		beg += 1;
	}
	__syncthreads();
	do {
		if(lane_id == 0) data_in = s_qp[beg];
		else data_in = __shfl(data_out, lane_id - 1, WARP);
		if(check_active(data_in)) {
			if(lane_id == LANE) printf("[%ld, %ld] ", data_in.h, data_in.e);
			data_out = data_in;
			reset(&data_in);
			beg += 1;
		} else {
			if(lane_id == LANE) printf("[nothing] ");
		}
	} while(beg < qlen);
}

int main(int argc, char *argv[])
{
	int qlen;
	printf("Input qlen = ");
	scanf("%d", &qlen);

	int8_t *h_qp, *d_qp;
	h_qp = (int8_t*)malloc(sizeof(eh_t) * (qlen + 1)));
	int k = 0;
	for(int i = 0; i <= qlen; i++) {
		h_qp[i].h = h_qp[i].e = i;
	}

	gpuErrchk(hipMalloc(&d_qp, sizeof(eh_t) * (qlen + 1)));
	gpuErrchk(hipMemcpy(d_qp, h_qp, sizeof(eh_t) * (qlen + 1), hipMemcpyHostToDevice));


	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventRecord(start, 0);

	func_0<<<1, WARP, qlen + 1>>>(qlen, d_qp);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	fprintf(stderr, "[M::%s] Kernel executed in %f ms\n" , __func__, elapsedTime);

	return 0;
	
}
