/*
 ============================================================================
 Name        : cuda_lock.cu
 Author      : vuongp
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA thread wide lock
 ============================================================================
 */

#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", \
			hipGetErrorString(code), file, line);
		if (abort) 
			exit(code);
	}
}

__device__ int mLock = 0;

__global__ void func(unsigned int *comm) {
	bool blocked = true;	
	while(blocked) {
	    if(0 == atomicCAS(&mLock, 0, 1)) {

	    	comm += 1;
	    	printf("Block Id = %d, Thread Id = %d, comm = %u", blockIdx.x, threadIdx.x, *comm);
	        atomicExch(&mLock, 0);
	        blocked = false;
	    }
	}
}
int main(void)
{
	unsigned int *d_comm;
	gpuErrchk(hipMalloc(&d_comm, sizeof(unsigned int)));
	gpuErrchk(hipMemset(d_comm, 0, sizeof(unsigned int)));
	func<<<2, 32>>>(d_comm);
	gpuErrchk(hipPeekAtLastError());
	return 0;
}


